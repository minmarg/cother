#include "hip/hip_runtime.h"
/***************************************************************************
 *   Copyright (C) 2013-2021 Mindaugas Margelevicius                       *
 *   Institute of Biotechnology, Vilnius University                        *
 ***************************************************************************/

#include "liblib/mybase.h"
#include "libmycu/cucom/cucommon.h"
#include "libmycu/cucom/warpscan.cuh"
#include "libmycu/cupro/SerializedScoresSM.cuh"
#include "libmycu/cupro/SerializedScoresAttr.h"
#include "libmycu/cupro/SerializedDstMatchScores.cuh"
#include "libmycu/cupro/PM2DVectorFields.h"
#include "libmycu/cupro/CuBatchProcessing_com.cuh"
#include "CuBatchScoreMatrix_com.h"
#include "CuBatchScoreMatrix_ddms2s.cuh"

// =========================================================================

// NOTE: parameters are passed to the device via constant memory and are 
// limited to 4 KB
// 
// device functions for computing batch score matrices;
// ddms2scores, serialized distance match scores and the map between 
// accumulated match scores and translated scores;
// dmsattr, attributes of ddms2scores
// ddmswgt, weight of scoring DDMS2S scores
// dstsegm, indivisible fragment of consecutive distance values
// nqyposs, number of query positions to process
// ndb1poss, number of cached db profile positions to process
// ndbCposs, number of new db profile positions to process
// dbxpad, number of padded positions for memory alignment
// querposoffset, offset from the origin of the device buffers allocated for 
// queries;
// bdb1posoffset, offset from the origin of the device buffers allocated for 
// cached db profile data;
// bdbCposoffset, offset from the origin of the device buffers allocated for 
// new (read) db profile data;
//

// #define CUSCO_DDMS2S_SMEMUnroll2x_TESTPRINT 1
// -------------------------------------------------------------------------
// CalcSM_DDMS2S_SMEMUnroll2x: device code for calculating pairwise 
// distance distribution match scores;
// matrix using shared memory and twofold unrolling along the x axis;
// each thread block processes two matrix blocks actually; 
// query profile data remains the same for these two spatially prallel 
// blocks;
// NOTE: it uses more shared memory but allows for greater occupancy;
// although bank conflicts cannot be avoided (due to random acces to the 
// SMEM), using SMEM garantees much greater efficiency in comparison to 
// other types of memory;
// NOTE: output pointers should be aligned!
// NOTE: results add to outscores and rewrite outmodscores;
// 
__global__ void CalcSM_DDMS2S_SMEMUnroll2x(
    CUBSM_TYPE* __restrict__ ddms2scores,
    SerializedDstMatchScoresAttr dmsattr,
    const float ddmswgt,
    const int dstsegm,
    uint nqyposs, uint ndb1poss, uint ndbCposs, uint dbxpad,
    uint querposoffset, uint bdb1posoffset, uint bdbCposoffset,
    CUBSM_TYPE* __restrict__ outscores,
    CUBSM_TYPE* __restrict__ outmodscores )
{
    extern __shared__ CUBSM_TYPE ddms2sCache[];//cached serialized scores
    //
    __shared__ FPTYPE 
            qrenoCache;//cache for query ENO
//             dbenoCache[2][SMINIT_2DCACHE_DIM];//cache for the ENOs of db profiles
    __shared__ FPTYPE 
            qrdstCache[ptr2DNoDstsPerPos][SMINIT_2DCACHE_DIM],//cache for a tile of IR Dsts at query positions
            dbdstCache[ptr2DNoDstsPerPos][SMINIT_2DCACHE_DIM],//cache for a tile of IR Dsts at db profile positions
            db2dstCache[ptr2DNoDstsPerPos][SMINIT_2DCACHE_DIM];//cache for a tile of IR Dsts at sec. db profile positions
    //
    uint blockbeg_y = blockIdx.y * blockDim.y;
    uint row = blockbeg_y + threadIdx.y;
    const uint col = blockIdx.x * blockDim.x * 2 + threadIdx.x;//logical column
    const uint col2 = col + blockDim.x;//logical column
    //physical indices:
    uint db1pos;
    uint db1pos2;
    uint dbfldsndx;
    uint dbfldsndx2;
    if( col < ndb1poss ) {  db1pos = col + bdb1posoffset;
                            dbfldsndx = pmv2DTotFlds;
    } else {                db1pos = col - ndb1poss + bdbCposoffset;//jump to section ndbCposs
                            dbfldsndx = TIMES2(pmv2DTotFlds);
    }
    if( col2 < ndb1poss ) { db1pos2 = col2 + bdb1posoffset;
                            dbfldsndx2 = pmv2DTotFlds;
    } else {                db1pos2 = col2 - ndb1poss + bdbCposoffset;//jump to section ndbCposs
                            dbfldsndx2 = TIMES2(pmv2DTotFlds);
    }
    //
    uint dbpronr;
    //
    //{{CACHE SERIALIZED SCORES using coalescent access (reuse registers)
    //NOTE: IMPORTANT: total number of entries (attr.ntotents_) is 
    // assumed to be less than the block size;
    //if this is not the case, uncomment the for loop for n-fold caching!
    dbpronr = threadIdx.y * blockDim.x + threadIdx.x;
    if( dbpronr < dmsattr.ntotents_ )
        ddms2sCache[dbpronr] = ddms2scores[dbpronr];
    //for( dbpronr += blockDim.x; dbpronr < dmsattr.ntotents_; dbpronr += blockDim.x )
    //    ddms2sCache[dbpronr] = ddms2scores[dbpronr];
    //for( ; dbpronr < ssattr.ntotents_ + cvattr.ntotents_; dbpronr += blockDim.x )
    //    ddms2sCache[dbpronr] = ddms2scores[dbpronr-dmsattr.ntotents_];
    //}}
    //



    //for comments, see the CalcSMInit... kernels
    //
    //read query ENO
    //NOTE: valid when processing one query at a time
    if( threadIdx.y < 1 && threadIdx.x < 1 ) {
        uint qpronr = ((INTYPE*)(dc_pm2dvfields_[pmv2DAddrPro]))[blockbeg_y+querposoffset];
        //read only one element per block (blockDim.y x blockDim.x)
        qrenoCache = ((FPTYPE*)(dc_pm2dvfields_[pps2DENO]))[qpronr];
    }

    //cache query IR distance values
    CacheIRDistanceValues(
        qrdstCache, pmv2DDDvalues,
        blockbeg_y + querposoffset + threadIdx.x,
        blockbeg_y + threadIdx.x < nqyposs );

    //cache db profile IR distance values
    CacheIRDistanceValues(
        dbdstCache, dbfldsndx + pmv2DDDvalues,
        db1pos,
        col < (ndb1poss + ndbCposs) );

    //cache second db profile's IR distance values
    CacheIRDistanceValues(
        db2dstCache, dbfldsndx2 + pmv2DDDvalues,
        db1pos2,
        col2 < (ndb1poss + ndbCposs) );



    //the warp reads data written by other warps, sync
    __syncthreads();

    if( nqyposs <= row || (ndb1poss + ndbCposs) <= col )
        //NOTE: NO sync after the exit of some of the threads
        return;

//     //reuse registers
//     dbpronr = (col2 < (ndb1poss + ndbCposs));

    float score1;

    CalcDDMS2ScoreSMEM(
        ddms2sCache,
        dmsattr.card_,
        ddmswgt,
        dstsegm,
        qrenoCache,
        qrdstCache,
        dbdstCache,
        score1);

    row = row * (ndb1poss + ndbCposs + dbxpad);

    //perform coalescent write of scores
    //atomicAdd is faster than += when we need coalescent write performed once
    if(score1) {
        atomicAdd( &outscores[row + col], score1 );
        //outmodscores[row + col] = score1 - CONSTCVSSHIFT * ddmswgt;
    }

// //     if( dbpronr/*col2 < (ndb1poss + ndbCposs)*/)
    if(col2 < (ndb1poss + ndbCposs)) {
        CalcDDMS2ScoreSMEM(
            ddms2sCache,
            dmsattr.card_,
            ddmswgt,
            dstsegm,
            qrenoCache,
            qrdstCache,
            db2dstCache,
            score1);

// //     if( dbpronr/*col2 < (ndb1poss + ndbCposs)*/) {
        if(score1) {
            atomicAdd( &outscores[row + col2], score1 );
            //outmodscores[row + col2] = score1 - CONSTCVSSHIFT * cvswgt;
        }
// //     }
    }

#ifdef CUSCO_DDMS2S_SMEMUnroll2x_TESTPRINT
    if(!blockIdx.x && !blockIdx.y && !threadIdx.x && !threadIdx.y)
        SerializedDstMatchScoresSM<CUBSM_TYPE,DMS_APPROXIMATE_SCORES>::PrintScores(ddms2sCache, dmsattr.card_);
    //
    int querypos = blockIdx.y * blockDim.y + threadIdx.y + querposoffset;
    dbpronr = ((INTYPE*)(dc_pm2dvfields_[dbfldsndx+pmv2DAddrPro]))[db1pos];
    uint qpronr = ((INTYPE*)(dc_pm2dvfields_[pmv2DAddrPro]))[querypos];
    uint dbpronr2 = 99999;
    MYASSERT( qrenoCache == ((FPTYPE*)(dc_pm2dvfields_[pps2DENO]))[qpronr], "Inconsistency.");
    int nqrdstvals = (int)qrdstCache[0][threadIdx.y];
    int ndbdstvals = (int)dbdstCache[0][threadIdx.x];
    int ndb2dstvals = -1;
    for( int i = 1; i <= nqrdstvals; i++ )
        MYASSERT( qrdstCache[i][threadIdx.y] == ((FPTYPE*)(dc_pm2dvfields_[pmv2DDDvalues+i]))[querypos], "Inconsistency.");
    for( int i = 1; i <= ndbdstvals; i++ )
        MYASSERT( dbdstCache[i][threadIdx.x] == ((FPTYPE*)(dc_pm2dvfields_[dbfldsndx+pmv2DDDvalues+i]))[db1pos], "Inconsistency.");
    if( col2 < (ndb1poss + ndbCposs)) {
        dbpronr2 = ((INTYPE*)(dc_pm2dvfields_[dbfldsndx2+pmv2DAddrPro]))[db1pos2];
        ndb2dstvals = (int)db2dstCache[0][threadIdx.x];
        for( int i = 1; i <= ndb2dstvals; i++ ) {
            MYASSERT( db2dstCache[i][threadIdx.x] == 
                ((FPTYPE*)(dc_pm2dvfields_[dbfldsndx2+pmv2DDDvalues+i]))[db1pos2], "Inconsistency.");
        }
    }
    if((dbpronr == CUSCO_DDMS2S_SMEMUnroll2x_TESTPRINT || dbpronr2 == CUSCO_DDMS2S_SMEMUnroll2x_TESTPRINT) &&
        blockIdx.x==14 && blockIdx.y==1 && threadIdx.x==0 && threadIdx.y==0) {
        printf(" Query %d (ENO %.1f):\n",qpronr,qrenoCache);
        for(int q=0; q < blockDim.y; q++) {
            nqrdstvals = (int)qrdstCache[0][q];
            printf(" %4d D: %d", querypos+q+1, nqrdstvals);
            for( int i = 1; i <= nqrdstvals; i++ ) printf(" %.0f",qrdstCache[i][q]);
            printf("\n");
        }
        if(dbpronr == CUSCO_DDMS2S_SMEMUnroll2x_TESTPRINT) {
            printf(" Db pro %d:\n", dbpronr);
            for(int q=0; q < blockDim.x; q++) {
                ndbdstvals = (int)dbdstCache[0][q];
                printf(" %4d D: %d", db1pos+q+1, ndbdstvals);
                for( int i = 1; i <= ndbdstvals; i++ ) printf(" %.0f",dbdstCache[i][q]);
                printf("\n");
            }
        } else {
            printf(" Db2 pro %d:\n", dbpronr2);
            for(int q=0; q < blockDim.x; q++) {
                ndb2dstvals = (int)db2dstCache[0][q];
                printf(" %4d D: %d", db1pos2+q+1, ndb2dstvals);
                for( int i = 1; i <= ndb2dstvals; i++ ) printf(" %.0f",db2dstCache[i][q]);
                printf("\n");
            }
        }
    }
#endif
}
